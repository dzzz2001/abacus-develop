#include "kernels/cuda/cuda_tools.cuh"
#include "module_base/ylm.h"
#include "gint_rho_gpu.h"
#include "gint_tools.h"
#include "kernels/cuda/gint_rho.cuh"

#include <omp.h>

namespace GintKernel
{

void gint_gamma_rho_gpu(const hamilt::HContainer<double>* dm,
                        const double* ylmcoef_now,
                        const double dr,
                        const double* rcut,
                        const Grid_Technique& gridt,
                        const UnitCell& ucell,
                        double* rho)
{
    int dev_id = base_device::information::set_device_by_rank();
    // checkCuda(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

    const int nbzp = gridt.nbzp;
    const int nczp =nbzp * gridt.bz;
    const int num_mcell_on_proc = nczp * gridt.ncx * gridt.ncy;
    const int lgd = gridt.lgd;
    const int max_atom = gridt.max_atom;
    const int num_streams = gridt.nstreams;
    const int max_atom_per_bcell = max_atom * gridt.bxyz;
    const int max_atom_per_z = max_atom * nbzp;
    const int max_phi_per_z = max_atom_per_bcell * nbzp * ucell.nwmax;
    const int max_atompair_per_z = max_atom * max_atom * nbzp;

    std::vector<hipStream_t> streams(num_streams);
    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamCreate(&streams[i]));
    }

    Cuda_Mem_Wrapper<double> dr_part(max_atom_per_z * 3, num_streams, true);
    Cuda_Mem_Wrapper<uint8_t> atoms_type(max_atom_per_z, num_streams, true);
    // The first number in every group of two represents the number of atoms on that bigcell.
    // The second number represents the cumulative number of atoms up to that bigcell.
    Cuda_Mem_Wrapper<int> atoms_num_info(2 * nbzp, num_streams, true);

    Cuda_Mem_Wrapper<double> psi(max_phi_per_z, num_streams, false);
    Cuda_Mem_Wrapper<double> psi_dm(max_phi_per_z, num_streams, false);

    Cuda_Mem_Wrapper<double> gemm_alpha(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_m(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_n(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_k(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_lda(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldb(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldc(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_A(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_B(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_C(max_atompair_per_z, num_streams, true);
    
    Cuda_Mem_Wrapper<double> rho_g(num_mcell_on_proc, 1, false);
    Cuda_Mem_Wrapper<double*> dot_product(nbzp * gridt.bxyz, num_streams, true);

    Cuda_Mem_Wrapper<double> dm_matrix(lgd * lgd, 1, true);
    // retrieve the density matrix on the host
    for (int iat1 = 0; iat1 < ucell.nat; iat1++)
    {
        for (int iat2 = 0; iat2 < ucell.nat; iat2++)
        {
            const int it1 = ucell.iat2it[iat1];
            const int it2 = ucell.iat2it[iat2];
            const int lo1 = gridt.trace_lo[ucell.itiaiw2iwt(it1, ucell.iat2ia[iat1], 0)];
            const int lo2 = gridt.trace_lo[ucell.itiaiw2iwt(it2, ucell.iat2ia[iat2], 0)];

            hamilt::AtomPair<double>* tmp_ap = dm->find_pair(iat1, iat2);
            int orb_index = 0;
            if (tmp_ap == NULL)
            {
                continue;
            }
            for (int orb_i = 0; orb_i < tmp_ap->get_row_size(); orb_i++)
            {
                for (int orb_j = 0; orb_j < tmp_ap->get_col_size(); orb_j++)
                {
                    dm_matrix.get_host_pointer()[(lo1 + orb_i) * lgd + (lo2 + orb_j)]
                        = tmp_ap->get_pointer(0)[orb_index];
                    orb_index++;
                }
            }
        }
    }
    dm_matrix.copy_host_to_device_sync();

// calculate the rho for every nbzp bigcells
#pragma omp parallel for num_threads(num_streams) collapse(2)
    for (int i = 0; i < gridt.nbx; i++)
    {
        for (int j = 0; j < gridt.nby; j++)
        {
            // 20240620 Note that it must be set again here because 
            // cuda's device is not safe in a multi-threaded environment.

            checkCuda(hipSetDevice(dev_id));
            // get stream id
            const int sid = omp_get_thread_num();

            int max_m = 0;
            int max_n = 0;
            int atom_pair_num = 0;
            int atoms_per_z = 0;
            const int grid_index_ij = i * gridt.nby * nbzp + j * nbzp;

            // generate GPU tasks, including the calculation of psir, matrix
            // multiplication, and dot product
            gtask_rho(gridt,
                      grid_index_ij,
                      ucell,
                      dr_part.get_host_pointer(sid),
                      atoms_type.get_host_pointer(sid),
                      atoms_num_info.get_host_pointer(sid),
                      atoms_per_z);

            alloc_mult_dot_rho(
                gridt,
                ucell,
                grid_index_ij,
                max_atom,
                lgd,
                nczp,
                atoms_num_info.get_host_pointer(sid),
                psi.get_device_pointer(sid),
                psi_dm.get_device_pointer(sid),
                dm_matrix.get_device_pointer(),
                gemm_alpha.get_host_pointer(sid),
                gemm_m.get_host_pointer(sid),
                gemm_n.get_host_pointer(sid),
                gemm_k.get_host_pointer(sid),
                gemm_lda.get_host_pointer(sid),
                gemm_ldb.get_host_pointer(sid),
                gemm_ldc.get_host_pointer(sid),
                gemm_A.get_host_pointer(sid),
                gemm_B.get_host_pointer(sid),
                gemm_C.get_host_pointer(sid),
                max_m,
                max_n,
                atom_pair_num,
                rho_g.get_device_pointer(),
                dot_product.get_host_pointer(sid));
            
            dr_part.copy_host_to_device_async(streams[sid], sid, atoms_per_z * 3);
            atoms_type.copy_host_to_device_async(streams[sid], sid, atoms_per_z);
            atoms_num_info.copy_host_to_device_async(streams[sid], sid);

            gemm_alpha.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_m.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_n.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_k.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_lda.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldb.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldc.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_A.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_B.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_C.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            dot_product.copy_host_to_device_async(streams[sid], sid);
            
            psi.memset_device_async(streams[sid], sid, 0);
            psi_dm.memset_device_async(streams[sid], sid, 0);

            // Launching kernel to calculate psi
            dim3 grid_psi(nbzp, gridt.bxyz);
            dim3 block_psi(64);
            get_psi<<<grid_psi, block_psi, 0, streams[sid]>>>(
                gridt.ylmcoef_g,
                dr,
                gridt.bxyz,
                ucell.nwmax,
                max_atom,
                gridt.atom_nwl_g,
                gridt.atom_new_g,
                gridt.atom_ylm_g,
                gridt.atom_nw_g,
                gridt.rcut_g,
                gridt.nr_max,
                gridt.psi_u_g,
                gridt.mcell_pos_g,
                dr_part.get_device_pointer(sid),
                atoms_type.get_device_pointer(sid),
                atoms_num_info.get_device_pointer(sid),
                psi.get_device_pointer(sid));
            checkCudaLastError();

            // Performing matrix multiplication alpha * mat_dm * mat_psir
            gridt.fastest_matrix_mul(max_m,
                                     max_n,
                                     gemm_m.get_device_pointer(sid),
                                     gemm_n.get_device_pointer(sid),
                                     gemm_k.get_device_pointer(sid),
                                     gemm_A.get_device_pointer(sid),
                                     gemm_lda.get_device_pointer(sid),
                                     gemm_B.get_device_pointer(sid),
                                     gemm_ldb.get_device_pointer(sid),
                                     gemm_C.get_device_pointer(sid),
                                     gemm_ldc.get_device_pointer(sid),
                                     atom_pair_num,
                                     streams[sid],
                                     gemm_alpha.get_device_pointer(sid));
            checkCudaLastError();

            // Launching kernel to calculate dot product psir * psir_dm
            // if warpSize is not eauql to 32, the psir_dot kernel should be modified
            dim3 grid_dot(nbzp, gridt.bxyz);
            dim3 block_dot(64); 
            psir_dot<<<grid_dot, block_dot, sizeof(double) * 32, streams[sid]>>>(
                gridt.bxyz,
                ucell.nwmax,
                atoms_num_info.get_device_pointer(sid),
                psi.get_device_pointer(sid),
                psi_dm.get_device_pointer(sid),
                dot_product.get_device_pointer(sid));
            checkCudaLastError();
            checkCuda(hipStreamSynchronize(streams[sid]));
        }
    }

    // Copy rho from device to host
    checkCuda(hipMemcpy(rho,
                         rho_g.get_device_pointer(),
                         num_mcell_on_proc * sizeof(double),
                         hipMemcpyDeviceToHost));

    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamDestroy(streams[i]));
    }
}

void gint_k_rho_gpu(const hamilt::HContainer<double>* dm,
                        const double* ylmcoef_now,
                        const double dr,
                        const double* rcut,
                        const Grid_Technique& gridt,
                        const UnitCell& ucell,
                        double* rho)
{
    int dev_id = base_device::information::set_device_by_rank();
    // checkCuda(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

    const int nbzp = gridt.nbzp;
    const int nczp =nbzp * gridt.bz;
    const int num_mcell_on_proc = nczp * gridt.ncx * gridt.ncy;
    const int lgd = gridt.lgd;
    const int max_atom = gridt.max_atom;
    const int num_streams = gridt.nstreams;
    const int max_atom_per_bcell = max_atom * gridt.bxyz;
    const int max_atom_per_z = max_atom * nbzp;
    const int max_phi_per_z = max_atom_per_bcell * nbzp * ucell.nwmax;
    const int max_atompair_per_z = max_atom * max_atom * nbzp;
    std::vector<bool> is_malloced(gridt.nnrg, false);

    std::vector<hipStream_t> streams(num_streams);
    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamCreate(&streams[i]));
    }

    Cuda_Mem_Wrapper<double> dr_part(max_atom_per_z * 3, num_streams, true);
    Cuda_Mem_Wrapper<uint8_t> atoms_type(max_atom_per_z, num_streams, true);
    // The first number in every group of two represents the number of atoms on that bigcell.
    // The second number represents the cumulative number of atoms up to that bigcell.
    Cuda_Mem_Wrapper<int> atoms_num_info(2 * nbzp, num_streams, true);

    Cuda_Mem_Wrapper<double> psi(max_phi_per_z, num_streams, false);
    Cuda_Mem_Wrapper<double> psi_dm(max_phi_per_z, num_streams, false);

    Cuda_Mem_Wrapper<double> gemm_alpha(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_m(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_n(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_k(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_lda(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldb(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldc(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_A(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_B(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_C(max_atompair_per_z, num_streams, true);
    
    Cuda_Mem_Wrapper<double> rho_g(num_mcell_on_proc, 1, false);
    Cuda_Mem_Wrapper<double*> dot_product(nbzp * gridt.bxyz, num_streams, true);

    Cuda_Mem_Wrapper<double> dm_matrix(gridt.nnrg, 1, false);

// calculate the rho for every nbzp bigcells
#pragma omp parallel for num_threads(num_streams) collapse(2)
    for (int i = 0; i < gridt.nbx; i++)
    {
        for (int j = 0; j < gridt.nby; j++)
        {
            // 20240620 Note that it must be set again here because 
            // cuda's device is not safe in a multi-threaded environment.

            checkCuda(hipSetDevice(dev_id));
            // get stream id
            const int sid = omp_get_thread_num();

            int max_m = 0;
            int max_n = 0;
            int atom_pair_num = 0;
            int atoms_per_z = 0;
            const int grid_index_ij = i * gridt.nby * nbzp + j * nbzp;

            // generate GPU tasks, including the calculation of psir, matrix
            // multiplication, and dot product
            gtask_rho(gridt,
                      grid_index_ij,
                      ucell,
                      dr_part.get_host_pointer(sid),
                      atoms_type.get_host_pointer(sid),
                      atoms_num_info.get_host_pointer(sid),
                      atoms_per_z);

            alloc_mult_dot_rho_k(
                gridt,
                ucell,
                dm,
                grid_index_ij,
                max_atom,
                lgd,
                nczp,
                atoms_num_info.get_host_pointer(sid),
                psi.get_device_pointer(sid),
                psi_dm.get_device_pointer(sid),
                dm_matrix.get_device_pointer(),
                gemm_alpha.get_host_pointer(sid),
                gemm_m.get_host_pointer(sid),
                gemm_n.get_host_pointer(sid),
                gemm_k.get_host_pointer(sid),
                gemm_lda.get_host_pointer(sid),
                gemm_ldb.get_host_pointer(sid),
                gemm_ldc.get_host_pointer(sid),
                gemm_A.get_host_pointer(sid),
                gemm_B.get_host_pointer(sid),
                gemm_C.get_host_pointer(sid),
                max_m,
                max_n,
                atom_pair_num,
                dm_matrix.get_device_pointer(),
                rho_g.get_device_pointer(),
                dot_product.get_host_pointer(sid),
                is_malloced,
                streams[sid]);
            
            dr_part.copy_host_to_device_async(streams[sid], sid, atoms_per_z * 3);
            atoms_type.copy_host_to_device_async(streams[sid], sid, atoms_per_z);
            atoms_num_info.copy_host_to_device_async(streams[sid], sid);

            gemm_alpha.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_m.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_n.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_k.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_lda.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldb.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldc.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_A.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_B.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_C.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            dot_product.copy_host_to_device_async(streams[sid], sid);
            
            psi.memset_device_async(streams[sid], sid, 0);
            psi_dm.memset_device_async(streams[sid], sid, 0);

            // Launching kernel to calculate psi
            dim3 grid_psi(nbzp, gridt.bxyz);
            dim3 block_psi(64);
            get_psi<<<grid_psi, block_psi, 0, streams[sid]>>>(
                gridt.ylmcoef_g,
                dr,
                gridt.bxyz,
                ucell.nwmax,
                max_atom,
                gridt.atom_nwl_g,
                gridt.atom_new_g,
                gridt.atom_ylm_g,
                gridt.atom_nw_g,
                gridt.rcut_g,
                gridt.nr_max,
                gridt.psi_u_g,
                gridt.mcell_pos_g,
                dr_part.get_device_pointer(sid),
                atoms_type.get_device_pointer(sid),
                atoms_num_info.get_device_pointer(sid),
                psi.get_device_pointer(sid));
            checkCudaLastError();

            // Performing matrix multiplication alpha * mat_dm * mat_psir
            gridt.fastest_matrix_mul(max_m,
                                     max_n,
                                     gemm_m.get_device_pointer(sid),
                                     gemm_n.get_device_pointer(sid),
                                     gemm_k.get_device_pointer(sid),
                                     gemm_A.get_device_pointer(sid),
                                     gemm_lda.get_device_pointer(sid),
                                     gemm_B.get_device_pointer(sid),
                                     gemm_ldb.get_device_pointer(sid),
                                     gemm_C.get_device_pointer(sid),
                                     gemm_ldc.get_device_pointer(sid),
                                     atom_pair_num,
                                     streams[sid],
                                     gemm_alpha.get_device_pointer(sid));
            checkCudaLastError();

            // Launching kernel to calculate dot product psir * psir_dm
            // if warpSize is not eauql to 32, the psir_dot kernel should be modified
            dim3 grid_dot(nbzp, gridt.bxyz);
            dim3 block_dot(64); 
            psir_dot<<<grid_dot, block_dot, sizeof(double) * 32, streams[sid]>>>(
                gridt.bxyz,
                ucell.nwmax,
                atoms_num_info.get_device_pointer(sid),
                psi.get_device_pointer(sid),
                psi_dm.get_device_pointer(sid),
                dot_product.get_device_pointer(sid));
            checkCudaLastError();
            checkCuda(hipStreamSynchronize(streams[sid]));
        }
    }

    // Copy rho from device to host
    checkCuda(hipMemcpy(rho,
                         rho_g.get_device_pointer(),
                         num_mcell_on_proc * sizeof(double),
                         hipMemcpyDeviceToHost));

    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamDestroy(streams[i]));
    }
    // printf("rho done");
    // dump_cuda_array_to_file(dm_matrix.get_device_pointer(), 1, gridt.nnrg, "dm_matrix");
    // exit(0);
    //     std::ofstream outFile("rho_new");
    //         if (!outFile.is_open())
    //         {
    //             std::cerr << "Failed to open file for writing." << std::endl;
    //         }
    //         for (int j = 0; j < num_mcell_on_proc; ++j)
    //         {
    //             outFile << "hight" << j <<  "   "
    //                     << rho[j] << std::endl;
    //         }
    //         outFile.close();
    //         exit(0);
}
} // namespace GintKernel
